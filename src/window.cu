#include "hip/hip_runtime.h"
#include "window.h"

void Window::draw_triangle(Point a, Point b, Point c) {
    a = point_to_screen(a);
    b = point_to_screen(b);
    c = point_to_screen(c);

    Point* d_a;
    Point* d_b;
    Point* d_c;

    void* d_pixels;
    float* d_depth_buffer;

    SDL_LockSurface(surface);

    hipMalloc(&d_a, sizeof(Point));
    hipMalloc(&d_b, sizeof(Point));
    hipMalloc(&d_c, sizeof(Point));
    hipMalloc(&d_pixels, surface->pitch * surface->h);
    hipMalloc(&d_depth_buffer, surface->w * surface->h * sizeof(int));

    hipMemcpy(d_a, &a, sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_c, &c, sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_pixels, surface->pixels, surface->pitch * surface->h, hipMemcpyHostToDevice);
    hipMemcpy(d_depth_buffer, this->depth_buffer, surface->w * surface->h * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y);
    draw_triangle_kernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, d_pixels, surface->pitch, surface->w, surface->h, d_depth_buffer);

    hipMemcpy(surface->pixels, d_pixels, surface->pitch * surface->h, hipMemcpyDeviceToHost);
    hipMemcpy(depth_buffer, d_depth_buffer, width * height * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_pixels);
    hipFree(d_depth_buffer);

    SDL_FreeSurface(surface);
}

__global__ void draw_triangle_kernel(Point* a, Point* b, Point* c, void* pixels, int pitch, int width, int height, float* depth_buffer) {
    float x = blockIdx.x * blockDim.x + threadIdx.x;
    float y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    Point p = {x, y, 0, 255, 255, 255};

    float area = edge_function(a, b, c);
    float w0 = edge_function(b, c, &p) / area;
    float w1 = edge_function(c, a, &p) / area;
    float w2 = edge_function(a, b, &p) / area;

    p.z = 1 / (w0 * a->z + w1 * b->z + w2 * c->z);

    float* depth = depth_buffer + (int) y * width + (int) x;

    if (p.z > *depth) {
        return;
    }

    if (w0 >= 0 && w1 >= 0 && w2 >= 0) {
        p.r = w0 * a->r + w1 * b->r + w2 * c->r * p.z;
        p.g = w0 * a->g + w1 * b->g + w2 * c->g * p.z;
        p.b = w0 * a->b + w1 * b->b + w2 * c->b * p.z;

        Uint8* pixel = (Uint8*) pixels;
        pixel += ((int) y * pitch) + ((int) x * sizeof(Uint32));

        pixel[2] = p.r;
        pixel[1] = p.g;
        pixel[0] = p.b;

        *depth = p.z;
    }
}

__device__ float edge_function(Point* a, Point* b, Point* c) {
    return (c->x - a->x) * (b->y - a->y) - (c->y - a->y) * (b->x - a->x);
}
